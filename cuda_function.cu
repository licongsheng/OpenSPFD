#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_complex.h"
#include "hipsparse.h"

#include <stdlib.h>
#include <stdio.h>
#include <complex>
#include <math.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <time.h>
#include <complex>
#include "common.h"
#include "hip/hip_complex.h"
#include "hipsparse.h"

using namespace std;

#define MAXTHREADS 1024
int max_threads = 1024;
int Num_devs = 0;

__global__ void SetData(float *Ax,float *Ay)
{
	int i = blockIdx.x *MAXTHREADS + threadIdx.x;
	Ax[i] = Ax[i] + Ay[i];//dlx/temp*Amp*1e-7;
	Ay[i] = 2;//dly/temp*Amp*1e-7;
}

__global__ void CalculateMagneticVector(float cx,float cy,float cz,float dlx,float dly,float dlz,
		float *x_axis,int nx,float *y_axis,int ny,float *z_axis,int nz,float Amp,
		float *Ax,float *Ay,float *Az,float *Bx,float *By,float *Bz)
{
	int N = nx*ny*nz;
	int i = blockIdx.x *MAXTHREADS + threadIdx.x;
	if(i < N)
	{
		int z = (int)((float)i / (float)(nx*ny));
		int res = i - z*nx*ny;
		int y = (int)((float)res / (float)nx);
		int x = res - y*nx;

		float dx = x_axis[x] - cx;
		float dy = y_axis[y] - cy;
		float dz = z_axis[z] - cz;
		float temp = sqrt(dx*dx+dy*dy+dz*dz);

		Ax[i] = Ax[i] + dlx/temp*Amp*1e-7;
		Ay[i] = Ay[i] + dly/temp*Amp*1e-7;
		Az[i] = Az[i] + dlz/temp*Amp*1e-7;

		temp = temp*temp*temp;
		Bx[i] = Bx[i] + (dly*dz - dlz*dy)/temp*Amp*1e-7;
		By[i] = By[i] + (dlz*dx - dlx*dz)/temp*Amp*1e-7;
		Bz[i] = Bz[i] + (dlx*dy - dly*dx)/temp*Amp*1e-7;
	}
	__syncthreads();
}


__global__ void ConstructSPFDA(hipComplex *eps,int dims[3],float delt[3],hipComplex *Sx)
{
	hipComplex Zero = make_hipFloatComplex(10,0);
	int i,j,k;
	long N = dims[0]*dims[1]*dims[2];
	int idx = blockIdx.x *MAXTHREADS + threadIdx.x;
	if(idx < N)
	{
		Sx[idx] = eps[idx];
		//Sy[idx] = eps[idx];
		//Sz[idx] = eps[idx];
		/*
		k = (int)((float)idx / (float)(dims[0]*dims[1]));
		int res = idx - k*dims[0]*dims[1];
		j = (int)((float)res / (float)dims[0]);
		i = res - j*dims[0];
		hipComplex yz_x = make_hipFloatComplex(delt[1]*delt[2] / delt[0],0);
		hipComplex h_yz_x = hipCmulf(make_hipFloatComplex(0.5,0),yz_x);
		hipComplex xz_y = make_hipFloatComplex(delt[0]*delt[2] / delt[1],0);
		hipComplex h_xz_y = hipCmulf(make_hipFloatComplex(0.5,0),xz_y);
		hipComplex xy_z = make_hipFloatComplex(delt[0]*delt[1] / delt[2],0);
		hipComplex h_xy_z = hipCmulf(make_hipFloatComplex(0.5,0),xy_z);

		if(i == 0)
			Sx[idx] = hipCmulf(eps[idx],yz_x);
		else
			Sx[idx] = hipCmulf(hipCaddf(eps[idx],eps[idx-1]),h_yz_x);

		if(j == 0)
			Sy[idx] = hipCmulf(eps[idx],xz_y);
		else
			Sy[idx] = hipCmulf(hipCaddf(eps[idx],eps[idx-dims[0]]),h_xz_y);

		if(k == 0)
			Sz[idx] = hipCmulf(eps[idx],xy_z);
		else
			Sz[idx] = hipCmulf(hipCaddf(eps[idx],eps[idx-dims[0]*dims[1]]),h_xy_z);
		 */
	}
}


__global__ void ConstructSPFDB(float *Ax,float *Ay,float *Az,int dims[3],float delt[3],hipComplex jw,
		hipComplex *Sx,hipComplex *Sy,hipComplex *Sz,hipComplex *SA)
{
	hipComplex Zero = make_hipFloatComplex(0,0);
	hipComplex Air = hipCmulf(make_hipFloatComplex(EPSILON,0),jw); //sigma = 0 S/m, epsilon = 1.0
	hipComplex dx = make_hipFloatComplex(delt[0],0);
	hipComplex dy = make_hipFloatComplex(delt[1],0);
	hipComplex dz = make_hipFloatComplex(delt[2],0);
	long N = dims[0]*dims[1]*dims[2];
	int idx = blockIdx.x *MAXTHREADS + threadIdx.x;
	if(idx < N)
	{
		int k = (int)((float)idx / (float)(dims[0]*dims[1]));
		int res = idx - k*dims[0]*dims[1];
		int j = (int)((float)res / (float)dims[0]);
		int i = res - j*dims[0];
		hipComplex Sx_n,Sx_p,Ax_n,Ax_p,Sy_n,Sy_p,Ay_n,Ay_p,Sz_n,Sz_p,Az_n,Az_p;
		if(i == 0)
		{
			Sx_n = Air;
			Sx_p = Sx[idx];
			Ax_n = Zero;
			Ax_p = make_hipFloatComplex(Ax[idx+1],0);
		}
		else if(i == dims[0]-1)
		{
			Sx_n = Sx[idx-1];
			Sx_p = Air;
			Ax_n = make_hipFloatComplex(Ax[idx-1],0);
			Ax_p = Zero;
		}
		else
		{
			Sx_n = Sx[idx-1];
			Sx_p = Sx[idx];
			Ax_n = make_hipFloatComplex(Ax[idx-1],0);
			Ax_p = make_hipFloatComplex(Ax[idx+1],0);
		}

		if(j == 0)
		{
			Sy_n = Air;
			Sy_p = Sy[idx];
			Ay_n = Zero;
			Ay_p = make_hipFloatComplex(Ay[idx+dims[0]],0);
		}
		else if(j == dims[1]-1)
		{
			Sy_n = Sy[idx-dims[0]];
			Sy_p = Air;
			Ay_n = make_hipFloatComplex(Ay[idx-dims[0]],0);
			Ay_p = Zero;
		}
		else
		{
			Sy_n = Sy[idx-dims[0]];
			Sy_p = Sy[idx];
			Ay_n = make_hipFloatComplex(Ay[idx-dims[0]],0);
			Ay_p = make_hipFloatComplex(Ay[idx+dims[0]],0);
		}

		if(k == 0)
		{
			Sz_n = Air;
			Sz_p = Sz[idx];
			Az_n = Zero;
			Az_p = make_hipFloatComplex(Az[idx+dims[0]*dims[1]],0);
		}
		else if(k == dims[2]-1)
		{
			Sz_n = Sz[idx-dims[0]*dims[1]];
			Sz_p = Air;
			Az_n = make_hipFloatComplex(Az[idx-dims[0]*dims[1]],0);
			Az_p = Zero;
		}
		else
		{
			Sz_n = Sz[idx-dims[0]*dims[1]];
			Sz_p = Sz[idx];
			Az_n = make_hipFloatComplex(Az[idx-dims[0]*dims[1]],0);
			Az_p = make_hipFloatComplex(Az[idx+dims[0]*dims[1]],0);
		}

		hipComplex temp = hipCaddf(hipCaddf(hipCsubf(hipCmulf(hipCmulf(Ax_p,Sx_p),dx),hipCmulf(hipCmulf(Ax_n,Sx_n),dx)),
				hipCsubf(hipCmulf(hipCmulf(Ay_p,Sy_p),dy),hipCmulf(hipCmulf(Ay_n,Sy_n),dy))),
				hipCsubf(hipCmulf(hipCmulf(Az_p,Sz_p),dz),hipCmulf(hipCmulf(Az_n,Sz_n),dz)));
		SA[idx] = hipCmulf(jw,temp);
	}
}


__global__ void cdotcKernel(hipComplex *x, hipComplex *y, int L)
{
	int i = blockIdx.x *MAXTHREADS + threadIdx.x;
	if(i < L)
	{
		hipComplex xc = make_hipComplex(x[i].x, -x[i].y);
		y[i] = hipComplexDoubleToFloat(hipCmul(hipComplexFloatToDouble(xc), hipComplexFloatToDouble(y[i])));
	}
}

__global__ void dotcKernel(hipComplex *x, hipComplex *y, hipComplex *z, int L)
{
	int i = blockIdx.x *MAXTHREADS + threadIdx.x;
	if (i < L)
	{
		hipComplex xc = make_hipComplex(x[i].x, -x[i].y);
		z[i] = hipComplexDoubleToFloat(hipCmul(hipComplexFloatToDouble(xc), hipComplexFloatToDouble(y[i])));
	}
}

__global__ void vecAddKernel(hipComplex alpha, hipComplex *x, hipComplex beta, hipComplex *y, int L)
{
	int i = blockIdx.x *MAXTHREADS + threadIdx.x;
	if (i < L)
	{
		y[i] = hipComplexDoubleToFloat(hipCadd(hipCmul(hipComplexFloatToDouble(x[i]), hipComplexFloatToDouble(alpha)) ,
			hipCmul(hipComplexFloatToDouble(y[i]), hipComplexFloatToDouble(beta))));
	}
}

__global__ void vecCaddKernel(hipComplex alpha, hipComplex *x, hipComplex beta, hipComplex *y, hipComplex *z, int L)
{
	int i = blockIdx.x *MAXTHREADS + threadIdx.x;
	if (i < L)
	{
		z[i] = hipComplexDoubleToFloat(hipCadd(hipCmul(hipComplexFloatToDouble(x[i]), hipComplexFloatToDouble(alpha)),
			hipCmul(hipComplexFloatToDouble(y[i]), hipComplexFloatToDouble(beta))));
	}
}


__global__ void mvMulKernel(int *RowPtr, int *ColIdx, hipComplex*A, hipComplex *x, hipComplex *b, int N, long nnz)
{
	int i = blockIdx.x *MAXTHREADS + threadIdx.x;
	if (i < N)
	{
		b[i] = make_hipComplex(0,0);
		for(int j= RowPtr[i];j<RowPtr[i+1];j++)
			b[i] = hipComplexDoubleToFloat(hipCadd(hipComplexFloatToDouble(b[i]),
				hipCmul(hipComplexFloatToDouble(A[j]), hipComplexFloatToDouble(x[ColIdx[j]]))));
	}
}



extern "C"
{


/*
 *  Calculate Magnetic Vector and Flux ...
 * */
hipError_t CUDAAnalysisMagneticField(float *Rx,float *Ry,float *Rz,float *dLx,float *dLy,float *dLz,int N_seg,
		float *x_axis,int nx,float *y_axis,int ny,float *z_axis,int nz,float Amp,
		float *Ax,float *Ay,float *Az,float *Bx,float *By,float *Bz,int devid)
{
	hipError_t cudaStatus;
	float *x_axis_d = NULL;
	float *y_axis_d = NULL;
	float *z_axis_d = NULL;
	float *Ax_d = NULL;
	float *Ay_d = NULL;
	float *Az_d = NULL;
	float *Bx_d = NULL;
	float *By_d = NULL;
	float *Bz_d = NULL;
	int nblocks = 0;
	long N = nx*ny*nz;
	int i=0;

	cudaStatus = hipSetDevice(devid);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!\n  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&x_axis_d, nx * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA malloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&y_axis_d, ny * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA malloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&z_axis_d, nz * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA malloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(x_axis_d, x_axis, nx * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'x_axis' failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(y_axis_d, y_axis, ny * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'y_axis' failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(z_axis_d, z_axis, nz * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'z_axis' failed!\n");
		goto Error;
	}

	//////////////////////////////////////////////////////////////////////////
	cudaStatus = hipMalloc((void**)&Ax_d, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA malloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&Ay_d, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA malloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&Az_d, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA malloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(Ax_d, Ax, N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'Ax' failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(Ay_d, Ay, N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'Ay' failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(Az_d, Az, N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'Az' failed!\n");
		goto Error;
	}

	//////////////////////////////////////////////////////////////////////////
	cudaStatus = hipMalloc((void**)&Bx_d, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA malloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&By_d, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA malloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&Bz_d, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA malloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(Bx_d, Bx, N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'Bx' failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(By_d, By, N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'By' failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(Bz_d, Bz, N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'Bz' failed!\n");
		goto Error;
	}

	nblocks = (int)ceil((double)N / (double)max_threads);
	/////////////////////////////////////////////////////////////////////////////////////
	for(i=0;i<N_seg;i++)
	{
		cout<< "Process for coil segmentation @ " <<i << "/" << N_seg << ". block = "<< nblocks << ", threads = " << MAXTHREADS <<endl;
		CalculateMagneticVector <<<nblocks, MAXTHREADS >>> (Rx[i],Ry[i],Rz[i],dLx[i],dLy[i],dLz[i],
			x_axis_d,nx,y_axis_d,ny,z_axis_d,nz,Amp,Ax_d,Ay_d,Az_d,Bx_d,By_d,Bz_d);
	}

	/////////////////////////////////////////////////////////////////////////////////////

	cudaStatus = hipMemcpy(Ax, Ax_d, N * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(Ay, Ay_d, N * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(Az, Az_d, N * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}


	cudaStatus = hipMemcpy(Bx, Bx_d, N * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(By, By_d, N * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(Bz, Bz_d, N * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}
	/////////////////////////////////////////////////////////////////////////////////////
Error:
	if(x_axis_d) hipFree(x_axis_d);
	if(y_axis_d) hipFree(y_axis_d);
	if(z_axis_d) hipFree(z_axis_d);
	if(Ax_d) hipFree(Ax_d);
	if(Ay_d) hipFree(Ay_d);
	if(Az_d) hipFree(Az_d);
	if(Bx_d) hipFree(Bx_d);
	if(By_d) hipFree(By_d);
	if(Bz_d) hipFree(Bz_d);

	return cudaStatus;
}


hipError_t CUDAAnalysisSxyz(complex<float> *cEps,float *Ax,float *Ay,float *Az,float f,
		complex<float> *Sx,complex<float> *Sy,complex<float> *Sz,
		complex<float> *SA,int dims[3],float spacing[3],int devid)
{
	hipError_t cudaStatus;
	hipComplex *cEps_d = NULL;
	float *Ax_d = NULL;
	float *Ay_d = NULL;
	float *Az_d = NULL;
	hipComplex *Sx_d = NULL;
	hipComplex *Sy_d = NULL;
	hipComplex *Sz_d = NULL;
	hipComplex *SA_d = NULL;
	hipComplex jw = make_hipComplex(0,2*M_PI*f);
	int nblocks = 0;
	long N = dims[0]*dims[1]*dims[2];

	cudaStatus = hipSetDevice(devid);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!\n  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	/////////////////////////////////////////////////////////////////////////////
	cudaStatus= hipMalloc((void**)&cEps_d, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA malloc failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(cEps_d, cEps, N * sizeof(hipComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'complex epsilon' failed!\n");
		goto Error;
	}

	/////////////////////////////////////////////////////////////////////////////
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&Ax_d, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA malloc failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&Ay_d, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA malloc failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&Az_d, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA malloc failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(Ax_d, Ax, N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'Ax' failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(Ay_d, Ay, N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'Ay' failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(Az_d, Az, N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'Az' failed!\n");
		goto Error;
	}

	/////////////////////////////////////////////////////////////////////////////
	cudaStatus= hipMalloc((void**)&Sx_d, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA malloc failed!\n");
		goto Error;
	}
	cudaStatus= hipMalloc((void**)&Sy_d, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA malloc failed!\n");
		goto Error;
	}
	cudaStatus= hipMalloc((void**)&Sz_d, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA malloc failed!\n");
		goto Error;
	}
	cudaStatus= hipMalloc((void**)&SA_d, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA malloc failed!\n");
		goto Error;
	}
	cudaStatus = hipMemset((void *)Sx_d, 0, sizeof(Sx_d[0]) * N);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'Sx' failed!\n");
		goto Error;
	}
	cudaStatus = hipMemset((void *)Sy_d, 0, sizeof(Sy_d[0]) * N);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'Sy' failed!\n");
		goto Error;
	}
	cudaStatus = hipMemset((void *)Sz_d, 0, sizeof(Sz_d[0]) * N);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'Sz' failed!\n");
		goto Error;
	}
	cudaStatus = hipMemset((void *)SA_d, 0, sizeof(SA_d[0]) * N);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'SA' failed!\n");
		goto Error;
	}
	////////////////////////////////////////////////////////////////////////////////////////
	nblocks = (int)ceil((double)N / (double)max_threads);
	cout<<"Constructing SPFD .... "<<endl;
	ConstructSPFDA <<<nblocks, MAXTHREADS >>> (cEps_d,dims,spacing,Sx_d);
	//ConstructSPFDB <<<nblocks, MAXTHREADS >>> (Ax_d,Ay_d,Az_d,dims,spacing,jw,
	//				Sx_d,Sy_d,Sz_d,SA_d);
	////////////////////////////////////////////////////////////////////////////////////////
	cudaStatus = hipMemcpy(Sx, Sx_d, N * sizeof(hipComplex), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'Sx_d' failed! %s\n",hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipMemcpy(Sy, Sy_d, N * sizeof(hipComplex), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'Sy_d' failed! %s\n",hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipMemcpy(Sz, Sz_d, N * sizeof(hipComplex), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'Sz_d' failed! %s\n",hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipMemcpy(SA, SA_d, N * sizeof(hipComplex), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy 'SA_d' failed! %s\n",hipGetErrorString(cudaStatus));
		goto Error;
	}
	//////////////////////////////////////////////////////////////////////////////////////////
Error:
	if(cEps_d) hipFree(cEps_d);
	if(Ax_d) hipFree(Ax_d);
	if(Ay_d) hipFree(Ay_d);
	if(Az_d) hipFree(Az_d);
	if(Sx_d) hipFree(Sx_d);
	if(Sy_d) hipFree(Sy_d);
	if(Sz_d) hipFree(Sz_d);
	if(SA_d) hipFree(SA_d);
	return cudaStatus;
}


float CudaSquaredNorm(hipComplex *x, long N)
{
	complex<float> *x_host = new complex<float>[N];
	hipMemcpy(x_host, x, N * sizeof(hipComplex), hipMemcpyDeviceToHost);

	float sum = 0;
	for (int i = 0; i < N; i++)
		sum = sum + x_host[i].real()*x_host[i].real() + x_host[i].imag()*x_host[i].imag();

	delete[]x_host;
	return sum;
}

complex<float> CudaSum(hipComplex *x, long N)
{
	complex<float> *x_host = new complex<float>[N];
	hipMemcpy(x_host, x, N * sizeof(hipComplex), hipMemcpyDeviceToHost);

	complex<float> sum = 0;
	for (int i = 0; i < N; i++)
		sum = sum + x_host[i];

	delete[]x_host;
	return sum;
}

hipError_t CudaVectorCdotc(complex<float> *x_host, complex<float> *y_host, complex<float> &dot, int L,int devid)
{
	hipError_t cudaStatus;
	hipComplex *x_dev = NULL;
	hipComplex *y_dev = NULL;
	complex<float> *z_host = NULL;
	int nblocks = 0;

	cudaStatus = hipSetDevice(devid);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&x_dev, L * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&y_dev, L * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(x_dev, x_host, L * sizeof(hipComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(y_dev, y_host, L * sizeof(hipComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	nblocks = (int)ceil((double)L / (double)max_threads);
	cdotcKernel <<<nblocks, MAXTHREADS >>> (x_dev,y_dev,L);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "MatrixMULTVector Kernal launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	z_host = new complex<float>[L];
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(z_host, y_dev, L * sizeof(hipComplex), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	dot = 0;
	for (int i = 0; i < L; i++)
		dot = dot + z_host[i];


Error:
	if (x_dev) hipFree(x_dev);
	if (y_dev) hipFree(y_dev);
	if (z_host) delete[]z_host;
	return cudaStatus;
}

hipError_t CudaVectorCadd(complex<float> alpha, complex<float> *x_host, complex<float> beta, complex<float> *y_host, int L, int devid)
{
	hipError_t cudaStatus;
	hipComplex *x_dev = NULL;
	hipComplex *y_dev = NULL;
	hipComplex cu_alpha = make_hipComplex(alpha.real(), alpha.imag());
	hipComplex cu_beta = make_hipComplex(beta.real(), beta.imag());
	int nblocks = 0;

	cudaStatus = hipSetDevice(devid);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&x_dev, L * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&y_dev, L * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(x_dev, x_host, L * sizeof(hipComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(y_dev, y_host, L * sizeof(hipComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	nblocks = (int)ceil((double)L / (double)max_threads);
	vecAddKernel <<<nblocks, MAXTHREADS >>> (cu_alpha,x_dev, cu_beta, y_dev, L);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "MatrixMULTVector Kernal launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(y_host, y_dev, L * sizeof(hipComplex), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	if (x_dev) hipFree(x_dev);
	if (y_dev) hipFree(y_dev);
	return cudaStatus;
}

hipError_t CudamvMul(int *RowPtr_host, int *ColIdx_host, complex<float> *A_host, complex<float> *x_host, complex<float> *b_host, int N,long nnz, int devid)
{
	hipError_t cudaStatus;
	int *RowPtr_dev = NULL;
	int *ColIdx_dev = NULL;
	hipComplex *A_dev = NULL;
	hipComplex *X_dev = NULL;
	hipComplex *b_dev = NULL;
	int nblocks = 0;

	cudaStatus = hipSetDevice(devid);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&RowPtr_dev, (N+1) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&ColIdx_dev, nnz * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&A_dev, nnz * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&X_dev, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&b_dev, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	// Copy input vectors from host memory to GPU buffers.

	cudaStatus = hipMemcpy(RowPtr_dev, RowPtr_host, (N+1) * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(ColIdx_dev, ColIdx_host, nnz * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(A_dev, A_host, nnz * sizeof(hipComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(X_dev, x_host, N * sizeof(hipComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	nblocks = (int)ceil((double)N / (double)max_threads);
	mvMulKernel << <nblocks, MAXTHREADS >> > (RowPtr_dev, ColIdx_dev, A_dev, X_dev, b_dev, N,nnz);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "MatrixMULTVector Kernal launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(b_host, b_dev, N * sizeof(hipComplex), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
Error:
	if (RowPtr_dev) hipFree(RowPtr_dev);
	if (ColIdx_dev) hipFree(ColIdx_dev);
	if (A_dev) hipFree(A_dev);
	if (X_dev) hipFree(X_dev);
	if (b_dev) hipFree(b_dev);
	return cudaStatus;
}


hipError_t GPUBICGSTAB(int *RowPtr_host, int *ColIdx_host, complex<float> *A_host, complex<float> *b_host, complex<float> *x_host, int N, long nnz, vector<float> residuals, int max_steps, float tol,int devid)
{
	hipError_t cudaStatus;
	int *RowPtr_dev = NULL;
	int *ColIdx_dev = NULL;
	hipComplex *A_dev = NULL;
	hipComplex *X_dev = NULL;
	hipComplex *b_dev = NULL;

	hipComplex *r = NULL;
	hipComplex *rh = NULL;
	hipComplex *v = NULL;
	hipComplex *p = NULL;
	hipComplex *temp = NULL;
	hipComplex *h = NULL;
	hipComplex *s = NULL;
	hipComplex *t = NULL;
	float nrmr, nrmr0;
	hipComplex rho0, rho, alpha, w, beta;
	float err = 1000;
	int interation = 0;
	hipComplex zero = make_hipComplex(0.0, 0.0);
	hipComplex one = make_hipComplex(1.0, 0.0);
	hipComplex none = make_hipComplex(-1.0, 0.0);
	int nblocks = (int)ceil((double)N / (double)max_threads);

	////////////////////////////////////////////////////////////////////////////
	cudaStatus = hipSetDevice(devid);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}


	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&RowPtr_dev, (N + 1) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&ColIdx_dev, nnz * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&A_dev, nnz * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&X_dev, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&b_dev, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.

	cudaStatus = hipMemcpy(RowPtr_dev, RowPtr_host, (N + 1) * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(ColIdx_dev, ColIdx_host, nnz * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(A_dev, A_host, nnz * sizeof(hipComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(b_dev, b_host, N * sizeof(hipComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(X_dev, x_host, N * sizeof(hipComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	////////////////////////////////////////////////////////////////////////////////////////////////////
	cudaStatus = hipMalloc((void**)&r, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&rh, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&v, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&p, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&h, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&s, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&t, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&temp, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(r, b_host, N * sizeof(hipComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(rh, b_host, N * sizeof(hipComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemset((void *)v, 0, sizeof(v[0]) * N);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}
	cudaStatus = hipMemset((void *)p, 0, sizeof(p[0]) * N);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}
	cudaStatus = hipMemset((void *)h, 0, sizeof(h[0]) * N);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}
	cudaStatus = hipMemset((void *)s, 0, sizeof(s[0]) * N);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}
	cudaStatus = hipMemset((void *)t, 0, sizeof(t[0]) * N);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}
	cudaStatus = hipMemset((void *)temp, 0, sizeof(temp[0]) * N);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}
	////////////////////////////////////////////////////////////////////////////////////////

	rho0 = alpha = w = one;
	nrmr0 = CudaSquaredNorm(r, N);
	cout << "BICGStab on GPU start iterating ..." << err << tol << interation << max_steps  <<endl;
	while (err > tol && interation < max_steps)
	{
		//1. ρi = (r̂0, ri−1)
		//cout << "BICGStab @ step 1"<< endl;
		dotcKernel <<<nblocks, MAXTHREADS >> > (rh, r,temp,N);
		complex<float> rho_host = CudaSum(temp, N);
		rho = make_hipComplex(rho_host.real(), rho_host.imag());
		hipMemset((void *)temp, 0, sizeof(temp[0]) * N);

		//2. β = (ρi/ρi−1)(α/ωi−1)
		//cout << "BICGStab @ step 2" << endl;
		beta = hipComplexDoubleToFloat(hipCmul(hipCdiv(hipComplexFloatToDouble(rho), hipComplexFloatToDouble(rho0)),
			hipCdiv(hipComplexFloatToDouble(alpha), hipComplexFloatToDouble(w))));
		rho0 = rho;

		//3. pi = ri−1 + β(pi−1 − ωi−1vi−1)
		//cout << "BICGStab @ step 3" << endl;
		hipComplex nw = make_hipComplex(-w.x,-w.y);
		vecCaddKernel << <nblocks, MAXTHREADS >> > (one, p, nw, v, temp, N);
		vecCaddKernel << <nblocks, MAXTHREADS >> > (one, r, beta, temp, p, N);
		hipMemset((void *)temp, 0, sizeof(temp[0]) * N);

		//4. vi = Api
		//cout << "BICGStab @ step 4" << endl;
		mvMulKernel << <nblocks, MAXTHREADS >> > (RowPtr_dev, ColIdx_dev, A_dev, p, v, N, nnz);

		//5. α = ρi/(r̂0, vi)
		//cout << "BICGStab @ step 5" << endl;
		dotcKernel << <nblocks, MAXTHREADS >> > (rh, v, temp, N);
		complex<float> rhv_host = CudaSum(temp, N);
		hipDoubleComplex rhv = make_hipDoubleComplex(rhv_host.real(), rhv_host.imag());
		hipMemset((void *)temp, 0, sizeof(temp[0]) * N);
		alpha = hipComplexDoubleToFloat(hipCdiv(hipComplexFloatToDouble(rho),rhv));

		//6. h = xi−1 + αpi
		//cout << "BICGStab @ step 6" << endl;
		vecCaddKernel << <nblocks, MAXTHREADS >> > (one, X_dev, alpha, p, h, N);
		//7. If h is accurate enough, then set xi = h and quit

		//8. s = ri−1 − αvi
		//cout << "BICGStab @ step 8" << endl;
		hipComplex nalpha = make_hipComplex(-alpha.x, -alpha.y);
		vecCaddKernel << <nblocks, MAXTHREADS >> > (one, r, nalpha, v, s, N);

		//9. t = As
		//cout << "BICGStab @ step 9" << endl;
		mvMulKernel << <nblocks, MAXTHREADS >> > (RowPtr_dev, ColIdx_dev, A_dev, s, t, N, nnz);

		//10. ωi = (t, s)/(t, t)
		//cout << "BICGStab @ step 10" << endl;
		//hipDoubleComplex tt = make_hipDoubleComplex(0, 0);
		//hipDoubleComplex ts = make_hipDoubleComplex(0, 0);
		dotcKernel << <nblocks, MAXTHREADS >> > (t, t, temp, N);
		complex<float> tt_host = CudaSum(temp, N);
		hipMemset((void *)temp, 0, sizeof(temp[0]) * N);

		dotcKernel << <nblocks, MAXTHREADS >> > (t, s, temp, N);
		complex<float> ts_host = CudaSum(temp, N);
		hipMemset((void *)temp, 0, sizeof(temp[0]) * N);
		complex<float> w_host = ts_host / tt_host;
		w = make_hipComplex(w_host.real(),w_host.imag());

		//11. xi = h + ωis
		//cout << "BICGStab @ step 11" << endl;
		vecCaddKernel << <nblocks, MAXTHREADS >> > (one, h, w, s, X_dev, N);

		//12. If xi is accurate enough, then quit

		//13. ri = s − ωit
		//cout << "BICGStab @ step 13" << endl;
		nw = make_hipComplex(-w.x, -w.y);
		vecCaddKernel << <nblocks, MAXTHREADS >> > (one, s, nw, t, r, N);

		nrmr = CudaSquaredNorm(r, N);

		err = sqrt(nrmr / nrmr0);
		cout << "BICGStab @ " << interation;
		cout.setf(ios::scientific, ios::floatfield);
		cout << ", Residual = " << err <<endl;
		residuals.push_back(err);
		interation++;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(x_host, X_dev, N * sizeof(hipComplex), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	if (RowPtr_dev) hipFree(RowPtr_dev);
	if (ColIdx_dev) hipFree(ColIdx_dev);
	if (A_dev) hipFree(A_dev);
	if (X_dev) hipFree(X_dev);
	if (b_dev) hipFree(b_dev);
	if (r) hipFree(r);
	if (rh) hipFree(rh);
	if (v) hipFree(v);
	if (p) hipFree(p);
	if (temp) hipFree(temp);
	if (h) hipFree(h);
	if (s) hipFree(s);
	if (t) hipFree(t);
	return cudaStatus;
}



void LowerTriSolver(complex<float> *A, complex<float> *b, complex<float> *X, int *R_ptr, int *C_idx, int N, long nnz)
{
	int i = 0;
	int j = 0;
	X[0] = b[0];

	for (i = 1; i < N; i++)
	{
		complex<float> s = 0;
		for (j = R_ptr[i]; j < R_ptr[i + 1]; j++)
			if (C_idx[j] < i)
				s = s + A[j] * X[C_idx[j]];
		X[i] = b[i] - s;
	}
}

void UpperTriSolver(complex<float> *A, complex<float> *b, complex<float> *X, int *R_ptr, int *C_idx, int N, long nnz)
{
	int i = 0;
	int j = 0;
	X[N - 1] = b[N - 1];

	for (i = N - 2; i >= 0; i--)
	{
		complex<float> s = 0;
		complex<float> dig = 1.0;
		for (j = R_ptr[i]; j < R_ptr[i + 1]; j++)
		{
			if (C_idx[j] == i)
				dig = A[j];
			if (C_idx[j] > i)
				s = s + A[j] * X[C_idx[j]];
		}

		X[i] = (b[i] - s) / dig;
	}
}



hipError_t GPUPBICGSTAB(int *RowPtr_host, int *ColIdx_host, complex<float> *A_host, complex<float> *A_ilu_host, complex<float> *b_host, complex<float> *x_host, int N, long nnz,vector<float> residuals, int max_steps, float tol, int devid, const char *logfile)
{
	hipEvent_t start, stop;
	float costtime;
	hipError_t cudaStatus;
	int *RowPtr_dev = NULL;
	int *ColIdx_dev = NULL;
	hipComplex *A_dev = NULL;
	hipComplex *iluA_dev = NULL;
	hipComplex *X_dev = NULL;
	hipComplex *b_dev = NULL;

	hipComplex *r = NULL;
	hipComplex *rh = NULL;
	hipComplex *v = NULL;
	hipComplex *p = NULL;
	hipComplex *y = NULL;
	hipComplex *temp = NULL;
	hipComplex *h = NULL;
	hipComplex *z = NULL;
	hipComplex *s = NULL;
	hipComplex *t = NULL;

	complex<float> *p_host = NULL;
	complex<float> *temp_host = NULL;
	complex<float> *y_host = NULL;

	complex<float> *s_host = NULL;
	complex<float> *z_host = NULL;
	clock_t cup_start, cup_stop;
	ofstream file;

	float nrmr, nrmr0;
	hipComplex rho0, rho, alpha, w, beta;
	float err = 1000;
	int interation = 0;
	hipComplex zero = make_hipComplex(0.0, 0.0);
	hipComplex one = make_hipComplex(1.0, 0.0);
	hipComplex none = make_hipComplex(-1.0, 0.0);
	int nblocks = (int)ceil((double)N / (double)max_threads);
	time_t tt = time(NULL);
	struct tm* current_time = localtime(&tt);

	////////////////////////////////////////////////////////////////////////////
	cudaStatus = hipSetDevice(devid);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&RowPtr_dev, (N + 1) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&ColIdx_dev, nnz * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&A_dev, nnz * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&iluA_dev, nnz * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&X_dev, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&b_dev, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.

	cudaStatus = hipMemcpy(RowPtr_dev, RowPtr_host, (N + 1) * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(ColIdx_dev, ColIdx_host, nnz * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(A_dev, A_host, nnz * sizeof(hipComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(iluA_dev, A_ilu_host, nnz * sizeof(hipComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(b_dev, b_host, N * sizeof(hipComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(X_dev, x_host, N * sizeof(hipComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	////////////////////////////////////////////////////////////////////////////////////////////////////
	cudaStatus = hipMalloc((void**)&r, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&rh, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&v, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&p, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&temp, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&y, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&h, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&s, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&z, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&t, N * sizeof(hipComplex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	/////////////////////////////////////////////////////////////////////////////////////
	cudaStatus = hipMemcpy(r, b_host, N * sizeof(hipComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(rh, b_host, N * sizeof(hipComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemset((void *)v, 0, sizeof(v[0]) * N);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}

	cudaStatus = hipMemset((void *)p, 0, sizeof(p[0]) * N);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}

	cudaStatus = hipMemset((void *)y, 0, sizeof(y[0]) * N);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}

	cudaStatus = hipMemset((void *)h, 0, sizeof(h[0]) * N);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}

	cudaStatus = hipMemset((void *)s, 0, sizeof(s[0]) * N);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}

	cudaStatus = hipMemset((void *)z, 0, sizeof(z[0]) * N);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}

	cudaStatus = hipMemset((void *)t, 0, sizeof(t[0]) * N);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}
	cudaStatus = hipMemset((void *)temp, 0, sizeof(temp[0]) * N);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}
	p_host = new complex<float>[N];
	memset(p_host,0,sizeof(p_host[0]) * N);
	temp_host = new complex<float>[N];
	memset(temp_host, 0, sizeof(temp_host[0]) * N);
	y_host = new complex<float>[N];
	memset(y_host, 0, sizeof(y_host[0]) * N);

	s_host = new complex<float>[N];
	memset(s_host, 0, sizeof(s_host[0]) * N);
	z_host = new complex<float>[N];
	memset(z_host, 0, sizeof(z_host[0]) * N);

	file.open(logfile);
	////////////////////////////////////////////////////////////////////////////////////
	rho0 = alpha = w = rho = one;
	nrmr0 = CudaSquaredNorm(r, N);
	cout << "BICGStab on GPU start iterating ..." << err << tol << interation << max_steps << endl;

	current_time = localtime(&tt);
	file << "current time is " << current_time->tm_hour<<":"<<current_time->tm_min<<":"<<current_time->tm_sec;

	//hipEventCreate(&start);
	//hipEventCreate(&stop);
	printf("Memory cost: %6.4f MB.\n",
			(float)ceil((11.0*N*(double)sizeof(hipComplex)+
					nnz*(double)sizeof(hipComplex)+
					(N+1)*(double)sizeof(int)+
					nnz*(double)sizeof(int))/1.0e6));
	while (err > tol && interation < max_steps)
	{
		rho0 = rho;
		//1. ρi = (r̂0, ri−1)
		//hipEventRecord(start,0);
		dotcKernel << <nblocks, MAXTHREADS >> > (rh, r, temp, N);
		//hipEventRecord(stop,0);
		//hipEventSynchronize(stop);
		//hipEventElapsedTime(&costtime,start,stop);
		//printf("CUDA Vector Inner Product spend %6.4f ms.\n",costtime);
		complex<float> rho_host = CudaSum(temp, N);
		rho = make_hipComplex(rho_host.real(), rho_host.imag());
		hipMemset((void *)temp, 0, sizeof(temp[0]) * N);

		//2. β = (ρi/ρi−1)(α/ωi−1)
		//cout << "BICGStab @ step 2" << endl;
		beta = hipComplexDoubleToFloat(hipCmul(
			hipCdiv(hipComplexFloatToDouble(rho), hipComplexFloatToDouble(rho0)),
			hipCdiv(hipComplexFloatToDouble(alpha), hipComplexFloatToDouble(w))));

		//3. pi = ri−1 + β(pi−1 − ωi−1vi−1)
		//cout << "BICGStab @ step 3" << endl;
		hipComplex nw = make_hipComplex(-w.x, -w.y);
		vecCaddKernel << <nblocks, MAXTHREADS >> > (one, p, nw, v, temp, N);
		vecCaddKernel << <nblocks, MAXTHREADS >> > (one, r, beta, temp, p, N);
		hipMemset((void *)temp, 0, sizeof(temp[0]) * N);

		//4. y = K−1pi
		// K=LU K=LU , p = L temp, temp = Uy
		//10.1 temp = L-1 p
		//hipEventRecord(start,0);
		//cup_start = clock();
		cudaStatus = hipMemcpy(p_host, p, N * sizeof(hipComplex), hipMemcpyDeviceToHost);
		memset(temp_host, 0, sizeof(temp_host[0]) * N);
		LowerTriSolver(A_ilu_host, p_host, temp_host, RowPtr_host, ColIdx_host, N, nnz);
		//10.2 y = U-1 temp
		UpperTriSolver(A_ilu_host, temp_host,y_host, RowPtr_host, ColIdx_host, N, nnz);
		cudaStatus = hipMemcpy(y, y_host, N * sizeof(hipComplex), hipMemcpyHostToDevice);
		//cup_stop = clock();
		//hipEventRecord(stop,0);
		//hipEventSynchronize(stop);
		//hipEventElapsedTime(&costtime,start,stop);
		//printf("CUDA Vector / SparseM spend %6.4f ms.\n",costtime);
		//printf("CPU Vector / SparseM spend %6.4f ms.\n",float(cup_stop - cup_start));
		//5. vi = Ay
		//hipEventRecord(start,0);
		mvMulKernel << <nblocks, MAXTHREADS >> > (RowPtr_dev, ColIdx_dev, A_dev, y, v, N, nnz);
		//hipEventRecord(stop,0);
		//hipEventSynchronize(stop);
		//hipEventElapsedTime(&costtime,start,stop);
		//printf("CUDA SparseM * Vector spend %6.4f ms.\n",costtime);

		//6. α = ρi/(r̂0, vi)
		dotcKernel << <nblocks, MAXTHREADS >> > (rh, v, temp, N);
		complex<float> rhv_host = CudaSum(temp, N);
		hipDoubleComplex rhv = make_hipDoubleComplex(rhv_host.real(), rhv_host.imag());
		hipMemset((void *)temp, 0, sizeof(temp[0]) * N);
		alpha = hipComplexDoubleToFloat(hipCdiv(hipComplexFloatToDouble(rho), rhv));

		//7. h = xi−1 + αy
		//vecCaddKernel << <nblocks, MAXTHREADS >> > (one, X_dev, alpha, p, h, N);

		//8. If h is accurate enough then xi = h and quit

		//9. s = ri−1 − αvi
		hipComplex nalpha = make_hipComplex(-alpha.x, -alpha.y);
		vecCaddKernel << <nblocks, MAXTHREADS >> > (one, r, nalpha, v, s, N);

		//10. z = K−1s
		//K=LU , s = L temp, temp = Uz
		//10.1 temp = L-1 s
		cudaStatus = hipMemcpy(s_host, s, N * sizeof(hipComplex), hipMemcpyDeviceToHost);
		memset(temp_host, 0, sizeof(temp_host[0]) * N);
		LowerTriSolver(A_ilu_host, s_host, temp_host, RowPtr_host, ColIdx_host, N, nnz);
		//10.2 z = U-1 temp
		UpperTriSolver(A_ilu_host, temp_host, z_host, RowPtr_host, ColIdx_host, N, nnz);
		cudaStatus = hipMemcpy(z, z_host, N * sizeof(hipComplex), hipMemcpyHostToDevice);

		//11. t = Az
		mvMulKernel << <nblocks, MAXTHREADS >> > (RowPtr_dev, ColIdx_dev, A_dev, z, t, N, nnz);

		//12. ωi = (K −1t, K −1s) / (K −1t, K −1t)
		dotcKernel << <nblocks, MAXTHREADS >> > (t, t, temp, N);
		complex<float> tt_host = CudaSum(temp, N);
		hipMemset((void *)temp, 0, sizeof(temp[0]) * N);
		if (tt_host.real() > 0)
		{
			dotcKernel << <nblocks, MAXTHREADS >> > (t, s, temp, N);
			complex<float> ts_host = CudaSum(temp, N);
			hipMemset((void *)temp, 0, sizeof(temp[0]) * N);
			complex<float> w_host = ts_host / tt_host;
			w = make_hipComplex(w_host.real(), w_host.imag());
		}
		else
			w = zero;

		//13. h = xi−1 + αy, xi = h + ωiz
		vecCaddKernel << <nblocks, MAXTHREADS >> > (one, X_dev, alpha, y, h, N);
		vecCaddKernel << <nblocks, MAXTHREADS >> > (one, h, w, z, X_dev, N);

		//14. If xi is accurate enough, then quit

		//15. ri = s − ωit
		//cout << "BICGStab @ step 13" << endl;
		nw = make_hipComplex(-w.x, -w.y);
		vecCaddKernel << <nblocks, MAXTHREADS >> > (one, s, nw, t, r, N);

		nrmr = CudaSquaredNorm(r, N);
		err = sqrt(nrmr / nrmr0);
		//err = nrmr / nrmr0;
		cout << "BICGStab @ " << interation;
		cout.setf(ios::scientific, ios::floatfield);
		cout << ", Residual = " << err << endl;
		file << "BICGStab @ " << interation;
		file.setf(ios::scientific, ios::floatfield);
		file << ", Residual = " << err << endl;
		residuals.push_back(err);
		interation++;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(x_host, X_dev, N * sizeof(hipComplex), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	tt = time(NULL);
	current_time = localtime(&tt);
	file << "current time is " << current_time->tm_hour << ":" << current_time->tm_min << ":" << current_time->tm_sec;
	file.close();

	//hipEventDestroy(start);
	//hipEventDestroy(stop);
Error:
	if (p_host) delete[]p_host;
	if (y_host) delete[]y_host;
	if (s_host) delete[]s_host;
	if (z_host) delete[]z_host;
	if (temp_host) delete[]temp_host;

	if (RowPtr_dev) hipFree(RowPtr_dev);
	if (ColIdx_dev) hipFree(ColIdx_dev);
	if (A_dev) hipFree(A_dev);
	if (iluA_dev) hipFree(iluA_dev);
	if (X_dev) hipFree(X_dev);
	if (b_dev) hipFree(b_dev);

	if (r) hipFree(r);
	if (rh) hipFree(rh);
	if (v) hipFree(v);
	if (p) hipFree(p);
	if (y) hipFree(y);
	if (temp) hipFree(temp);
	if (z) hipFree(z);
	if (h) hipFree(h);
	if (s) hipFree(s);
	if (t) hipFree(t);

	return cudaStatus;
}



}

